// Date March 28 2029
//Programer: Hemanta Bhattarai
// Progarm : To add two arrays and compare computation time in host and device

#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h> //for random numbers

#include <time.h>
#include <sys/time.h>

#define gpuErrchk(ans){ gpuAssert((ans),__FILE__, __LINE__);}


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if(code != hipSuccess)
	{
		fprintf(stderr, "GPUassert : %s %s %d\n", hipGetErrorString(code), file, line);
		if(abort) exit(code);
	}
}

const int threads_per_block = 128;



// device kernal
__global__ void dotProduct(float *A, float *B, float *D, int array_size)
{
	__shared__ float cache[threads_per_block];
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int cache_index = threadIdx.x;
	float temp = 0;
	while(i < array_size) 
	{

		temp += A[i] * B[i];
		i += blockDim.x * gridDim.x;  // each iteration will move the block-grid to access other element in matrix
	}

	cache[cache_index] = temp;
	__syncthreads(); //waits for all the threads to complete

	//for reductions, threadsPerBlock must be power of 2 due to following code
	int j = blockDim.x/2;
	while(j != 0){
		if (cache_index<j)
			cache[cache_index] += cache[cache_index + j];
		__syncthreads();
		j /=2;
		
	}

	if (cache_index == 0)
		D[blockIdx.x] = cache[0];	
}



int main()
{	
	// host function definition
	float get_random();
	
	//variable definition
	float *hA, *hB, *dA, *dB;
	float size_of_array, result_host, *result_device, *partial_result_from_device;
	
	

	
	//define size of array
	printf("Enter the size of array");
	scanf("%f",&size_of_array);
	float size = sizeof(float) * size_of_array;

	int blocks_per_grid = int(size_of_array/threads_per_block + 1);
	//memory allocation in host
	hA = (float*)malloc(size);
	hB = (float*)malloc(size);
	partial_result_from_device = (float*) malloc(sizeof(float) * blocks_per_grid);
	
	//memory allocation in device
	gpuErrchk(hipMalloc((void**)&dA,size));
	gpuErrchk(hipMalloc((void**)&dB,size));
	gpuErrchk(hipMalloc((void**)&result_device, blocks_per_grid * sizeof(float)));



	//array initilization 
	for(int i=0; i<size_of_array; ++i) hA[i] = get_random();
	for(int i=0; i<size_of_array; ++i) hB[i] = get_random();
	
	
	clock_t host_begin, host_end;
	//record begin of host computation
	host_begin = clock();

	//add vectors in host
	result_host = 0;
	for(int i=0; i<size_of_array; ++i) result_host += hA[i] * hB[i];

	//record end of host computation
	host_end = clock();


	
	clock_t device_begin, device_end;
	
	
	//record of device computation
	device_begin = clock();


	//copy host data to memory
	gpuErrchk(hipMemcpy(dA, hA, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dB, hB, size, hipMemcpyHostToDevice));
	
	//record start of device computation

	// dot product in device
	dotProduct<<<blocks_per_grid, threads_per_block>>>(dA, dB, result_device, size_of_array);
	
	gpuErrchk(hipDeviceSynchronize());



	//copy data from device to host
	gpuErrchk(hipMemcpy(partial_result_from_device, result_device, blocks_per_grid * sizeof(float), hipMemcpyDeviceToHost));

	float final_result = 0;
	for(int i=0; i< blocks_per_grid; i++) final_result += partial_result_from_device[i];

	//record end of device computation
	device_end = clock();
	

	double host_time, device_time;
	host_time = (double)((double)(host_end - host_begin)/(CLOCKS_PER_SEC));
	device_time = (double)((double)(device_end - device_begin)/(CLOCKS_PER_SEC));

	//print the time of host and device computation
	printf("Host computation time: %f\n",host_time);
	printf("Device computation time: %f\n",device_time);

	//display the devation of device and host result
	printf("The deviation of host and device result is %f\n",final_result - result_host);

	//free host memory
	free(hA);
	free(hB);

	//free device memory
	gpuErrchk(hipFree(dA));
	gpuErrchk(hipFree(dB));

	

}

//random number generator
float get_random()
{
	return rand() % 100 + 1;
}

