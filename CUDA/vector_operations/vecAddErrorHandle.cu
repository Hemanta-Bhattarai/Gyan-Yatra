// Date March 26 2029
//Programer: Hemanta Bhattarai
// Progarm : To add two arrays and compare computation time in host and device

#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h> //for random numbers

#include <time.h>
#include <sys/time.h>

#define gpuErrchk(ans){ gpuAssert((ans),__FILE__, __LINE__);}


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if(code != hipSuccess)
	{
		fprintf(stderr, "GPUassert : %s %s %d\n", hipGetErrorString(code), file, line);
		if(abort) exit(code);
	}
}





// device kernal
__global__ void vecAdd(int *A, int *B, int *C, int *D, int array_size)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	
	if(i < array_size) D[i] = A[i] + B[i] + C[i];
}



int main()
{	
	// host function definition
	int get_random();
	
	//variable definition
	int *hA, *hB, *hC, *hD, *hE, *dA, *dB, *dC, *dD;
	int size_of_array;
	
	

	
	//define size of array
	printf("Enter the size of array");
	scanf("%d",&size_of_array);
	dim3 grid(1024);
	dim3 block((size_of_array/grid.x)+1);	
	int size = sizeof(int) * size_of_array;

	//memory allocation in host
	hA = (int*)malloc(size);
	hB = (int*)malloc(size);
	hC = (int*)malloc(size);
	hD = (int*)malloc(size);
	hE = (int*)malloc(size);
	
	//memory allocation in device
	gpuErrchk(hipMalloc(&dA,size));
	gpuErrchk(hipMalloc(&dB,size));
	gpuErrchk(hipMalloc(&dC,size));
	gpuErrchk(hipMalloc(&dD,size));


	//array initilization 
	for(int i=0; i<size_of_array; ++i) hA[i] = get_random();
	for(int i=0; i<size_of_array; ++i) hB[i] = get_random();
	for(int i=0; i<size_of_array; ++i) hC[i] = get_random();
	
	
	clock_t host_begin, host_end;
	//record begin of host computation
	host_begin = clock();

	//add vectors in host
	for(int i=0; i<size_of_array; ++i) hE[i] = hA[i] + hB[i] + hC[i];

	//record end of host computation
	host_end = clock();


	
	clock_t device_begin, device_end;
	
	
	//record of device computation
	device_begin = clock();


	//copy host data to memory
	gpuErrchk(hipMemcpy(dA, hA, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dB, hB, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dC, hC, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dD, hD, size, hipMemcpyHostToDevice));
	
	//record start of device computation

	// add array in device
	vecAdd<<<block,grid>>>(dA, dB, dC, dD, size_of_array);


	//record end of device computation
	device_end = clock();


	//copy data from device to host
	gpuErrchk(hipMemcpy(hD, dD, size, hipMemcpyDeviceToHost));

	double host_time, device_time;
	host_time = (double)((double)(host_end - host_begin)/(CLOCKS_PER_SEC));
	device_time = (double)((double)(device_end - device_begin)/(CLOCKS_PER_SEC));

	//print the time of host and device computation
	printf("Host computation time: %f\n",host_time);
	printf("Device computation time: %f\n",device_time);

	//display the devation of device and host result
	int sum = 0;
	for(int i=0; i< size_of_array; ++i) sum += hE[i] - hD[i];
	printf("The deviation of host and device result is %d\n",sum);

	//free host memory
	free(hA);
	free(hB);
	free(hC);
	free(hD);
	free(hE);

	//free device memory
	gpuErrchk(hipFree(dA));
	gpuErrchk(hipFree(dB));
	gpuErrchk(hipFree(dC));
	gpuErrchk(hipFree(dD));

	

}

//random number generator
int get_random()
{
	return rand() % 100 + 1;
}

