//Date 1 April 2019
//Program: To multiply two matrices

#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h> //for random numbers

#include <time.h>
#include <sys/time.h>

#define gpuErrchk(ans){ gpuAssert((ans),__FILE__, __LINE__);}


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if(code != hipSuccess)
	{
		fprintf(stderr, "GPUassert : %s %s %d\n", hipGetErrorString(code), file, line);
		if(abort) exit(code);
	}
}




__global__ void matrix_multiply(int *A, int *B, int *C, int Am, int An, int Bn)
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	int sum = 0;
	if(i < Am && j < Bn)
	{
		for(int k=0; k<An; ++k)
		{
			sum += A[ i * An + k] * B[k * Bn + j];
		}

		C[i * Bn + j] = sum;
	}

}

int main()
{	
	// host function definition
	int get_random();
	int get_max(int, int);	
	//variable definition
	int *hA, *hB, *hC, *hD, *dA, *dB, *dC;
	int rows_A, columns_A, rows_B, columns_B;
	
	

	
	//define size of array
	do{
		printf("The number of columns of first matrix must \n be equal to number of rows of second matrix!!!\n");
		printf("Enter the rows and columns of A\n");
		scanf("%d",&rows_A);
		scanf("%d",&columns_A);

		printf("Enter the rows and columns of B\n");
		scanf("%d",&rows_B);
		scanf("%d",&columns_B);
	}while(columns_A != rows_B);


	
	dim3 block(get_max(rows_A,rows_B), get_max(columns_A, columns_B));	
	int size = sizeof(int) * get_max(rows_A * columns_A, rows_B * columns_B);

	//memory allocation in host
	hA = (int*)malloc(size);
	hB = (int*)malloc(size);
	hC = (int*)malloc(size);
	hD = (int*)malloc(size);
	
	//memory allocation in device
	gpuErrchk(hipMalloc((void**)&dA,size));
	gpuErrchk(hipMalloc((void**)&dB,size));
	gpuErrchk(hipMalloc((void**)&dC,size));


	//array initilization 
	for(int i=0; i<rows_A; ++i) 
	{
		for(int j=0; j< columns_A; ++j)
		{
			hA[i * columns_A + j] = get_random();

		}
	}
	
	for(int i=0; i<rows_B; ++i) 
	{
		for(int j=0; j< columns_B; ++j)
		{
			hB[i * columns_B + j] = get_random();

		}
	}	


	clock_t host_begin, host_end;
	//record begin of host computation
	host_begin = clock();

	//multiply matrix in host
	for(int i=0; i<rows_A; ++i) 
	{
		for(int j=0; j< columns_B; ++j)
		{
			int sum = 0;
			for(int k=0; k< columns_A; ++k) 
			{
			
				sum += hA[i * columns_A + k] * hB[k * columns_B + j];
				
			}
		
			hC[i * columns_B + j] = sum ;
			
		}
	}	


	//record end of host computation
	host_end = clock();


	
	clock_t device_begin, device_end;
	
	
	//record of device computation
	device_begin = clock();


	//copy host data to memory
	gpuErrchk(hipMemcpy(dA, hA, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dB, hB, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dC, hC, size, hipMemcpyHostToDevice));
	
	//record start of device computation

	// multiply matix in device
	matrix_multiply<<<1, block>>>(dA, dB, dC, rows_A, columns_A, columns_B );




	//copy data from device to host
	gpuErrchk(hipMemcpy(hD, dC, size, hipMemcpyDeviceToHost));

	//record end of device computation
	device_end = clock();

	double host_time, device_time;
	host_time = (double)((double)(host_end - host_begin)/(CLOCKS_PER_SEC));
	device_time = (double)((double)(device_end - device_begin)/(CLOCKS_PER_SEC));

	//print the time of host and device computation
	printf("++++++++++++++++++++++++++++++++++++++++++++++++++++++++");
	printf("\n\t\tHost computation time: %f\n",host_time);
	printf("\t\tDevice computation time: %f\n",device_time);
	printf("+++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");



	//-------------------------------------------------------------------------------
	/*
	// display element of A	
	printf("\n\n Matrix A\n\n");
	for(int i=0; i<rows_A; ++i) 
	{
		for(int j=0; j< columns_A; ++j)
		{
			printf("%d\t",hA[i * columns_A + j]);

		}
		printf("\n");
	}
	
	// display element of B	
	printf("\n\n Matrix B\n\n");
	for(int i=0; i<rows_B; ++i) 
	{
		for(int j=0; j< columns_B; ++j)
		{
			printf("%d\t",hA[i * columns_B + j]);

		}
		printf("\n");
	}
	
	// display element of AB	
	printf("\n\n Matrix AB\n\n");
	for(int i=0; i<rows_A; ++i) 
	{
		for(int j=0; j< columns_B; ++j)
		{
			printf("%d\t",hC[i * columns_B + j]);

		}
		printf("\n");
	}
	*/
	
	//display the devation of device and host result

	//--------------------------------------------------------------------------------------------




	int sum = 0;
	for(int i=0; i< rows_A; ++i) 
	{
		for(int j=0; j< columns_B; ++j)
		{
			sum += hD[i * columns_B + j] - hC[i * columns_B + j];
		}
	}
	printf("\nThe deviation of host and device result is %d\n",sum);

	
	
	//free host memory
	free(hA);
	free(hB);
	free(hC);
	free(hD);

	
	
	//free device memory
	gpuErrchk(hipFree(dA));
	gpuErrchk(hipFree(dB));
	gpuErrchk(hipFree(dC));

	

}

//random number generator
int get_random()
{
	return rand() % 10 + 1;
}


int get_max(int a, int b)
{
	return a >= b ? a : b;
}
