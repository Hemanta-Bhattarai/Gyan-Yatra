// Date March 26 2029
//Programer: Hemanta Bhattarai
// Progarm : To add two arrays




#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h> //for random numbers

// device kernal
__global__ void vecAdd(int *A, int *B, int *C)
{
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}



int main()
{	
	// host function definition
	int get_random();
	
	//variable definition
	int *hA, *hB, *hC, *dA, *dB, *dC;
	int size_of_array;
	
	
	//define size of array
	printf("Enter the size of array");
	scanf("%d",&size_of_array);
	
	int size = sizeof(int) * size_of_array;

	//memory allocation in host
	hA = (int*)malloc(size);
	hB = (int*)malloc(size);
	hC = (int*)malloc(size);
	
	//memory allocation in device
	hipMalloc(&dA,size);
	hipMalloc(&dB,size);
	hipMalloc(&dC,size);


	//array initilization 
	for(int i=0; i<size_of_array; ++i) hA[i] = get_random();
	for(int i=0; i<size_of_array; ++i) hB[i] = get_random();

	//copy host data to memory
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
	hipMemcpy(dC, hC, size, hipMemcpyHostToDevice);
	
	// add array in device
	vecAdd<<<1,size_of_array>>>(dA,dB,dC);

	//copy data from device to host
	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

	//display the result
	for(int i=0; i< size_of_array; ++i) printf("%d + %d = %d\n", hA[i], hB[i], hC[i]);

	//free host memory
	free(hA);
	free(hB);
	free(hC);

	//free device memory
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	

}

//random number generator
int get_random()
{
	return rand() % 100 + 1;
}

