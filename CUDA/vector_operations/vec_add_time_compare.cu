// Date March 26 2029
//Programer: Hemanta Bhattarai
// Progarm : To add two arrays and compare computation time in host and device




#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> //for random numbers

#include <time.h>
#include <sys/time.h>
// device kernal
__global__ void vecAdd(int *A, int *B, int *C)
{
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
}



int main()
{	
	// host function definition
	int get_random();
	
	//variable definition
	int *hA, *hB, *hC, *hD, *dA, *dB, *dC;
	int size_of_array;
	
	struct timeval begin, end;

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//define size of array
	printf("Enter the size of array");
	scanf("%d",&size_of_array);
	
	int size = sizeof(int) * size_of_array;

	//memory allocation in host
	hA = (int*)malloc(size);
	hB = (int*)malloc(size);
	hC = (int*)malloc(size);
	hD = (int*)malloc(size);
	
	//memory allocation in device
	hipMalloc(&dA,size);
	hipMalloc(&dB,size);
	hipMalloc(&dC,size);


	//array initilization 
	for(int i=0; i<size_of_array; ++i) hA[i] = get_random();
	for(int i=0; i<size_of_array; ++i) hB[i] = get_random();
	
	//record start of host computation
	gettimeofday(&begin,NULL);

	//add vectors in host
	for(int i=0; i<size_of_array; ++i) hD[i] = hA[i] + hB[i];

	//record end of host computation
	gettimeofday(&end,NULL);

	//copy host data to memory
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);
	hipMemcpy(dC, hC, size, hipMemcpyHostToDevice);
	
	//record start of device computation
	hipEventRecord(start,0);

	// add array in device
	vecAdd<<<1,size_of_array>>>(dA,dB,dC);


	//record end of device computation
	hipEventRecord(stop,0);

	float time_device;
	hipEventElapsedTime(&time_device, start, stop);

	//copy data from device to host
	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

	float time_host = 1e6* (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec);


	//print the time of host and device computation
	printf("Host computation time: %f\n",time_host);
	printf("Device computation time: %f\n",time_device);

	//display the devation of device and host result
	int sum = 0;
	for(int i=0; i< size_of_array; ++i) sum += hD[i] - hC[i];
	printf("The deviation of host and device result is %d\n",sum);

	//free host memory
	free(hA);
	free(hB);
	free(hC);
	free(hD);

	//free device memory
	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	

}

//random number generator
int get_random()
{
	return rand() % 100 + 1;
}

