// Date March 28 2029
//Programer: Hemanta Bhattarai
// Progarm : To add two arrays and compare computation time in host and device

#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h> //for random numbers

#include <time.h>
#include <sys/time.h>

#define gpuErrchk(ans){ gpuAssert((ans),__FILE__, __LINE__);}


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if(code != hipSuccess)
	{
		fprintf(stderr, "GPUassert : %s %s %d\n", hipGetErrorString(code), file, line);
		if(abort) exit(code);
	}
}





// device kernal
__global__ void vecAdd(float *A, float *B, float *C, float *D, int array_size)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	while(i < array_size) 
	{
		D[i] = A[i] + B[i] + C[i];
		i += blockDim.x * gridDim.x;  // each iteration will move the block-grid to access other element in matrix
	}
}



int main()
{	
	// host function definition
	float get_random();
	
	//variable definition
	float *hA, *hB, *hC, *hD, *dA, *dB, *dC;
        float *dD, *hE;
	float size_of_array;
	
	

	
	//define size of array
	printf("Enter the size of array");
	scanf("%f",&size_of_array);
	float size = sizeof(int) * size_of_array;

	//memory allocation in host
	hA = (float*)malloc(size);
	hB = (float*)malloc(size);
	hC = (float*)malloc(size);
	hD = (float*)malloc(size);
	hE = (float*)malloc(size);
	
	//memory allocation in device
	gpuErrchk(hipMalloc((void**)&dA,size));
	gpuErrchk(hipMalloc((void**)&dB,size));
	gpuErrchk(hipMalloc((void**)&dC,size));
	gpuErrchk(hipMalloc((void**)&dD,size));


	//array initilization 
	for(int i=0; i<size_of_array; ++i) hA[i] = get_random();
	for(int i=0; i<size_of_array; ++i) hB[i] = get_random();
	for(int i=0; i<size_of_array; ++i) hC[i] = get_random();
	
	
	clock_t host_begin, host_end;
	//record begin of host computation
	host_begin = clock();

	//add vectors in host
	for(int i=0; i<size_of_array; ++i) hE[i] = hA[i] + hB[i] + hC[i];

	//record end of host computation
	host_end = clock();


	
	clock_t device_begin, device_end;
	
	
	//record of device computation
	device_begin = clock();


	//copy host data to memory
	gpuErrchk(hipMemcpy(dA, hA, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dB, hB, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dC, hC, size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(dD, hD, size, hipMemcpyHostToDevice));
	
	//record start of device computation

	// add array in device
	vecAdd<<<128,128>>>(dA, dB, dC, dD, size_of_array);
	gpuErrchk(hipDeviceSynchronize());

	//record end of device computation
	device_end = clock();


	//copy data from device to host
	gpuErrchk(hipMemcpy(hD, dD, size, hipMemcpyDeviceToHost));

	double host_time, device_time;
	host_time = (double)((double)(host_end - host_begin)/(CLOCKS_PER_SEC));
	device_time = (double)((double)(device_end - device_begin)/(CLOCKS_PER_SEC));

	//print the time of host and device computation
	printf("Host computation time: %f\n",host_time);
	printf("Device computation time: %f\n",device_time);

	//display the devation of device and host result
	float sum = 0;
	for(int i=0; i< size_of_array; ++i) 
	{
		sum += hE[i] - hD[i];
	}
	printf("The deviation of host and device result is %f\n",sum);

	//free host memory
	free(hA);
	free(hB);
	free(hC);
	free(hD);
	free(hE);

	//free device memory
	gpuErrchk(hipFree(dA));
	gpuErrchk(hipFree(dB));
	gpuErrchk(hipFree(dC));
	gpuErrchk(hipFree(dD));

	

}

//random number generator
float get_random()
{
	return rand() % 100 + 1;
}

