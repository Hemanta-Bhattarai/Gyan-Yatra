
#include <hip/hip_runtime.h>
#include<stdio.h>


__global__ void print()
{
	int i = threadIdx.x;
	printf("%d\n",i);
}

int main()
{
	print<<<1,4>>>();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
	
}
