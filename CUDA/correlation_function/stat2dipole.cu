#include "hip/hip_runtime.h"




#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <sstream>

using namespace std;



#define gpuErrchk(ans){gpuAssert((ans), __FILE__, __LINE__);}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if(code != hipSuccess)
	{
		fprintf(stderr, "GPUassert : %s %s %d\n", hipGetErrorString(code), file, line);
		if(abort) exit(code);
	}
}




__global__ void dipoleCorrelation(double *px, double *py, double *pz, double *corr, int N)
{

  int tau = threadIdx.x + blockDim.x * blockIdx.x;
  double local_corr = 0;
	if(tau < N)
	{

  	for(int index = 0; index  < N - tau; ++index)
  	{

    	local_corr +=   px[index] * px[index + tau]
                  	+ py[index] * py[index + tau]
                  	+ pz[index] * pz[index + tau];
		

  	}
  	local_corr = local_corr/(N-tau);
  	corr[tau] = local_corr;
	}
	__syncthreads();
}

int main()
{
  string data, line, word;
  int pos(8);
  vector< double > dipole_x, dipole_y, dipole_z;
  vector< double > t;
  const string fileName = "Platinum_nanosphere_run2.stat";
  const string fileOut = "CorrfuncCuda.wcorr";

  ifstream file;

  //open file

  file.open(fileName,ios::in);
  if(!file)
  {
    cout<<"Error in opening file"<<endl;
    return -1;

  }

  while(!file.eof())
  {
   getline(file, line);
   int i = 0;
   stringstream is(line);

   while( is >> word )
   {
     if  (word.compare("#") == 0 || word.compare("##") == 0 ) break;
     if(i == 0) t.push_back(stod(word));
     if(i == pos)
     {
       dipole_x.push_back(stod(word));
     }
     if(i == pos + 1)
     {
       dipole_y.push_back(stod(word));
     }

     if(i == pos + 2)
     {
       dipole_z.push_back(stod(word));
     }
     i++;

   }
  }
cout<<"Dipole vector list created"<<endl;
//vector<double> dipole_corr, corr_time;

// calculation of co-orelation  function
ofstream outfile;
outfile.open(fileOut);
int N = dipole_x.size();
double *xcomp_dipole = &dipole_x[0];  //convert dipole_x vector to array
double *ycomp_dipole = &dipole_y[0];
double *zcomp_dipole = &dipole_z[0];

double *xcomp_dipole_d, *ycomp_dipole_d, *zcomp_dipole_d;
double *corr_h, *corr_d;
corr_h = (double*)malloc(N*sizeof(double));

double dt = t[1]-t[0];
cout<<"Finding the correlation funciton"<<endl;

gpuErrchk(hipMalloc((void**)&xcomp_dipole_d, N * sizeof(double)));
gpuErrchk(hipMalloc((void**)&ycomp_dipole_d, N * sizeof(double)));
gpuErrchk(hipMalloc((void**)&zcomp_dipole_d, N * sizeof(double)));
gpuErrchk(hipMalloc((void**)&corr_d, N * sizeof(double)));

/*
for(int index =0; index  < N ; ++index)
{
	printf("Index: %d Px: %e, Py: %e, Pz: %e\n",index,xcomp_dipole[index],ycomp_dipole[index],zcomp_dipole[index]);
	printf("Index: %d Px: %e, Py: %e, Pz: %e\n",index,dipole_x[index],dipole_y[index],dipole_z[index]);


}

*/


gpuErrchk(hipMemcpy(xcomp_dipole_d, xcomp_dipole, N * sizeof(double), hipMemcpyHostToDevice));
gpuErrchk(hipMemcpy(ycomp_dipole_d, ycomp_dipole, N * sizeof(double), hipMemcpyHostToDevice));
gpuErrchk(hipMemcpy(zcomp_dipole_d, zcomp_dipole, N * sizeof(double), hipMemcpyHostToDevice));
gpuErrchk(hipMemcpy(corr_d, corr_h, N * sizeof(double), hipMemcpyHostToDevice));


int number_of_blocks;
number_of_blocks = ( N/1024 ) + 1;

dipoleCorrelation<<<number_of_blocks,1024>>> (xcomp_dipole_d, ycomp_dipole_d, zcomp_dipole_d, corr_d, N);
gpuErrchk(hipDeviceSynchronize());

gpuErrchk(hipMemcpy(corr_h, corr_d, N * sizeof(double), hipMemcpyDeviceToHost));

outfile<<"## charge velocity autocorrelation function"<<endl;
outfile<<"# time(tau)\t wcorr"<<endl;
for(int count= 0; count < N ; ++count )
{
  outfile << t[count] << "\t" << corr_h[count]<<endl;
//	cout << t[count] << "\t" << corr_h[count]<<endl;
  //dipole_corr.push_back(local_corr/(length - tau));
  //corr_time.push_back(tau * dt);
}
outfile.close();



delete [] corr_h;
corr_h = NULL;

gpuErrchk(hipFree(corr_d));

}
